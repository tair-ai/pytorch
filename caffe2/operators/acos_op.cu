#include "hip/hip_runtime.h"
#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void AcosKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = acos(X[i]);
  }
}

template <typename T>
__global__ void AcosGradientKernel(const int N, const T* X, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = -dY[i] / sqrt(1 - X[i] * X[i]);
  }
}

struct AcosCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    AcosKernel<T>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           device_context->hip_stream()>>>(n, x, y);
    return;
  }
};

struct AcosGradientCUDAFunctor {
  template <typename T>
  inline void Run(
      const int n,
      const T* x,
      const T* dy,
      T* dx,
      HIPContext* device_context) {
    AcosGradientKernel<T>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           device_context->hip_stream()>>>(n, x, dy, dx);
    return;
  }
};

REGISTER_HIP_OPERATOR(
    Acos,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, AcosCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    AcosGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        WithoutBroadcast<AcosGradientCUDAFunctor>>);
} // namespace caffe2
